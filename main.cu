#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "run_utils.cuh"
#include <string>


int main(int argc, char* argv[]) {

    if (argc < 2) {
        printf("Too little parameter, you need to chose one kernel.\n");
        return 0;
    }

    constexpr int N = 1 << 11;
    constexpr int bytes = N * N * sizeof(int);
    constexpr float alpha = 1.5, beta = 0.4; // GEMM input parameters, C=α*AB+β*C

    printf("This is a gemm C = alpha * AB + beta * C, use square matrix for simplicity.\n");
    printf("Side length: %d \n", N);


    float *h_A, *h_B, *h_C;
    float *cpu_C; 
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);
    cpu_C = (float*)malloc(bytes);


    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    random_initialize(h_A, N);
    random_initialize(h_B, N);
    random_initialize(h_C, N);

    copy_matrix(cpu_C, h_C, N);
    cpu_gemm(h_A, h_B, cpu_C, N, alpha, beta);


    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice);
    

    int kernel = std::stoi(argv[1]);

    run_kernel(d_A, d_B, d_C, N, kernel, alpha, beta);

    

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    check_result(cpu_C, h_C, N);




    free(h_A);
    free(h_B);
    free(h_C);
    free(cpu_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    return 0;
}
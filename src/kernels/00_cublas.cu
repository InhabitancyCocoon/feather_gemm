#include "00_cublas.cuh"

void cublas_sgemm(const float* A, const float* B, float* C, const int N,
                              const float alpha, const float beta) {
    hipblasStatus_t cbStat;
    hipblasHandle_t cbHandle;
    cbStat = hipblasCreate(&cbHandle);
    cbStat = hipblasSgemm(cbHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, 
                         &alpha, B, N, A, N, &beta, C, N);
}